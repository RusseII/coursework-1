#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <cutil.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  out[i] = in1[i] + in2[i];
}

int main(int argc, char * * argv) {
  int inputLength1, inputLength2, outputLength;
  float * hostInput1, * hostInput2, * hostOutput, * deviceInput1, * deviceInput2, * deviceOutput, * expectedOutput;

  FILE * infile1, * infile2, * outfile;
  unsigned int generic, gpu, copy, compute, blog = 1;

  // Import host input data
  CUT_SAFE_CALL(cutCreateTimer( & generic));
  cutStartTimer(generic);
  if ((infile1 = fopen("input0.raw", "r")) == NULL) {
    printf("Cannot open input0.raw.\n");
    exit(EXIT_FAILURE);
  }
  if ((infile2 = fopen("input1.raw", "r")) == NULL) {
    printf("Cannot open input1.raw.\n");
    exit(EXIT_FAILURE);
  }
  fscanf(infile1, "%i", & inputLength1);
  hostInput1 = (float * ) malloc(sizeof(float) * inputLength1);
  for (int i = 0; i < inputLength1; i++)
    fscanf(infile1, "%f", & hostInput1[i]);
  fscanf(infile2, "%i", & inputLength2);
  hostInput2 = (float * ) malloc(sizeof(float) * inputLength2);
  for (int i = 0; i < inputLength2; i++)
    fscanf(infile2, "%f", & hostInput2[i]);
  fclose(infile1);
  fclose(infile2);
  hostOutput = (float * ) malloc(sizeof(float) * inputLength1);
  cutStopTimer(generic);
  printf("Importing data and creating memory on host: %f ms\n", cutGetTimerValue(generic));

  if (blog) printf("*** The input length is %i\n", inputLength1);

  CUT_SAFE_CALL(cutCreateTimer( & gpu));
  cutStartTimer(gpu);

  int size = sizeof(float) * inputLength1;
  hipMalloc((void * * ) & deviceInput1, size);
  hipMalloc((void * * ) & deviceInput2, size);
  hipMalloc((void * * ) & deviceOutput, size);

  cutStopTimer(gpu);
  printf("Allocating GPU memory: %f ms\n", cutGetTimerValue(gpu));

  cutDeleteTimer(gpu);
  CUT_SAFE_CALL(cutCreateTimer( & gpu));
  cutStartTimer(gpu);

  hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceOutput, hostOutput, size, hipMemcpyHostToDevice);

  cutStopTimer(gpu);
  printf("Copying input memory to the GPU: %f ms\n", cutGetTimerValue(gpu));

  dim3 gridDim(ceil(inputLength1 / 4), 1, 1);
  dim3 blockDim(inputLength1 / 4, 1, 1);

  if (blog) printf("*** Block dimension is %i\n", blockDim.x);
  if (blog) printf("*** Grid dimension is %i\n", gridDim.x);

  CUT_SAFE_CALL(cutCreateTimer( & compute));
  cutStartTimer(compute);

  vecAdd << < gridDim, blockDim >>> (deviceInput1, deviceInput2, deviceOutput, inputLength1);

  hipDeviceSynchronize();

  cutStopTimer(compute);
  printf("Performing CUDA computation: %f ms\n", cutGetTimerValue(compute));

  CUT_SAFE_CALL(cutCreateTimer( & copy));
  cutStartTimer(copy);

  hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);

  cutStopTimer(copy);
  printf("Copying output memory to the CPU: %f ms\n", cutGetTimerValue(copy));

  cutDeleteTimer(gpu);
  CUT_SAFE_CALL(cutCreateTimer( & gpu));
  cutStartTimer(gpu);

  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  cutStopTimer(gpu);
  printf("Freeing GPU Memory: %f ms\n", cutGetTimerValue(gpu));

  if ((outfile = fopen("output.raw", "r")) == NULL) {
    printf("Cannot open output.raw.\n");
    exit(EXIT_FAILURE);
  }
  fscanf(outfile, "%i", & outputLength);
  expectedOutput = (float * ) malloc(sizeof(float) * outputLength);
  for (int i = 0; i < outputLength; i++)
    fscanf(outfile, "%f", & expectedOutput[i]);
  fclose(outfile);
  int test = 1;
  int count = 0;
  for (int i = 0; i < outputLength; i++) {
    if (abs(expectedOutput[i] - hostOutput[i]) > 0.005) {
      test = 0;
      break;
    }
  }
  if (test) printf("Results correct.\n");
  else printf("Results incorrect.%d\n", count);

  cutDeleteTimer(generic);
  cutDeleteTimer(gpu);
  cutDeleteTimer(copy);
  cutDeleteTimer(compute);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(expectedOutput);

  return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
   printf("Hello world from block (%d,%d), thread (%d,%d,%d).\n",
           blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, threadIdx.z);

   __syncthreads();
}

int main(void) {
   int devID;
   hipDeviceProp_t p;

   hipGetDevice(&devID);
   hipGetDeviceProperties(&p, devID);
   printf("Running on device %d \"%s\" with capability %d.%d.\n",
           devID, p.name, p.major, p.minor);
   if (p.major < 2) {
      printf("Program incompatible with existing architecture; terminating.\n");
      return 1;
   }
   dim3 dimGrid(2,2);
   dim3 dimBlock(2,2,2);
   hello<<<dimGrid,dimBlock>>>();
   hipDeviceSynchronize();
   return 0;
}

